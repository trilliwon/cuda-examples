#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<string>
#include "gputimer.h"

#define TILE_WIDTH 2   /* set TILE_WIDTH 16 for the evaluation! */
#define MAXPOOL_INPUT_FILENAME "input.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpooling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int output_size = input_size / filter_size;

    // out of bound
    if (col >= output_size || row >= output_size) { return; }
    
    // 2D to 1D : (row, col) -> (row * N) + col
    float max_val = input[((row * filter_size) * input_size) + (col * filter_size)];

    for (int i = row * filter_size; i < row * filter_size + filter_size; i++) {
        for (int j = col * filter_size; j < col * filter_size + filter_size; j++) {
            // update max_val if needed
            max_val = fmaxf(max_val, input[(i * input_size) + j]);
        }
    }

    // assign max value
    output[(row * output_size) + col] = max_val;
}

int main(int argc, char **argv) {

    GpuTimer timer;
    if(argc < 2) {
        cout << "usage : " << argv[0] << " input_size filter_size alpha beta\n" << "example : " << argv[0] << " 100 2 0.5 0.8\n";
        return 1;
    }

    const int input_size = stoi(argv[1]);
    const int filter_size = stoi(argv[2]); // used for maxpooling
    const int maxpool_output_size = input_size/filter_size;

    // check input_size is power of 2
    if(input_size == 0 && (input_size & (input_size-1))){
        cout << "input_size must be power of 2\n";
        return 1;
    }

    if(filter_size == 0){
        cout << "filter_size cannot be 0\n";
        return 1;
    }

    float* maxpool_input = new float[input_size * input_size];
    
    // generate random array
    for (int i = 0; i < input_size*input_size; ++i) {
        maxpool_input[i] = rand() % 100;
    }
    
    // prints inputs for debugging.
    cout<<"filter size : "<<filter_size;
    cout<<"input size: " << input_size;
    cout<<"\n========== MAXPOOL_INPUT ==========\n";
    for (int i = 0; i < 100; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<maxpool_input[i]<<" ";
    }
    if (input_size > 10 ) {
        cout << ".....";
    }
    cout<<'\n';

    // set thread, block dimensions
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    const dim3 num_of_maxpool_blocks(maxpool_output_size/block_size.x+1, maxpool_output_size/block_size.y+1);

    // memory allocation for the device
    float *dev_mem_input, *maxpool_output;
    hipMalloc(&dev_mem_input, sizeof(float) * input_size * input_size);
    hipMalloc(&maxpool_output, sizeof(float) * maxpool_output_size * maxpool_output_size);

    // copy variable to device memory
    hipMemcpy(dev_mem_input, maxpool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
 
    timer.Start();

    // launch CUDA kernels
    // Then run maxpooling
    maxpool<<<num_of_maxpool_blocks, block_size>>>(dev_mem_input, maxpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
    timer.Stop();
    printf("Time elapsed = %g ms\n", timer.Elapsed());

    // allocate output buf in main memory
    float *maxpool_output_buf = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    
    // copy results from device to host
    hipMemcpy(maxpool_output_buf, maxpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    
    // prints the results
    cout<<"\n========== MAXPOOL OUTPUT ==========\n";
    for (int i = 0; i < maxpool_output_size * maxpool_output_size; ++i) {
        if(i%maxpool_output_size==0) cout<<"\n";
        cout<<maxpool_output_buf[i]<<" ";
    }
    cout<<'\n';

    hipFree(dev_mem_input);
    hipFree(maxpool_output);
    free(maxpool_output_buf);
	delete[] maxpool_input;
    return 0;
}
