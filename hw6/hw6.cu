/*
 * Find BLANK and replace your own code.
 * And submit report why do you replace the blank that way.
 */


#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<string>

#define TILE_WIDTH 2   /* set TILE_WIDTH 16 for the evaluation! */
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpolling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // out of bound

    // CHANGE
}

__global__ void gemm(float *a, float *b, float *c, const float alpha, const float beta, float *output, const int input_size){
    // a, b, c : input matrix address
    // alpha, beta : input constant
    // output : output buffer address
    // input_size : width, height of input matrix
    // all input, output matrices are vectorized

    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x,  by = blockIdx.y;

    int row = by*blockDim.y + ty;
    int col = bx*blockDim.x + tx;
    
    if(row>=input_size ||col>=input_size) { return; }
    
    // allocate 2D tiles in __shared__ memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    float result = 0;

    // make sure you handle the case when the matrix sizes are not
    // multiple of TILE_WIDTH!
    // loop over the tiles of the input in phases
    for(int p = 0; p < input_size/TILE_WIDTH; ++p){
        // CHANGE

        // You need to use __syncthreads() a few times
        // to synchronize the threads in a thread block.
    }

    // write out the result to output[row*input_size + col] 
    // CHANGE
}


int main(int argc, char **argv) {
    if(argc < 4) {
        cout << "usage : " << argv[0] << " input_size filter_size alpha beta\n" << "example : " << argv[0] << " 100 2 0.5 0.8\n";
        return 1;
    }
    const int input_size = stoi(argv[1]);
    const int filter_size = stoi(argv[2]); // used for maxpooling
    const float alpha = stof(argv[3]);
    const float beta = stof(argv[4]);
    const int maxpool_output_size = input_size/filter_size;

    // check input_size is power of 2
    if(input_size == 0 && (input_size & (input_size-1))){
        cout << "input_size must be power of 2\n";
        return 1;
    }

    if(filter_size == 0){
        cout << "filter_size cannot be 0\n";
        return 1;
    }

    float maxpool_input[input_size*input_size];
    float a[input_size*input_size];
    float b[input_size*input_size];
    float c[input_size*input_size];
    
    // read input matrices 
    ifstream input_in(MAXPOOL_INPUT_FILENAME);
    ifstream a_in(A_FILENAME);
    ifstream b_in(B_FILENAME);
    ifstream c_in(C_FILENAME);

    for (int i = 0; i < input_size*input_size; ++i) {
        input_in >> maxpool_input[i];
        a_in >> a[i];
        b_in >> b[i];
        c_in >> c[i];
    }
    
    // prints inputs for debugging.
    cout<<"filter size : "<<filter_size;
    cout<<"\n========== MAXPOOL_INPUT ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<maxpool_input[i]<<" ";
    }
    cout<<"\nalpha : "<<alpha<<'\n';
    cout<<"========== A ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<a[i]<<" ";
    }
    cout<<"\n========== B ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<b[i]<<" ";
    }
    cout<<"\nbeta : "<<beta<<'\n';
    cout<<"========== C ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<c[i]<<" ";
    }
    cout<<'\n';
       
    // set thread, block dimensions
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    const dim3 num_of_maxpool_blocks(maxpool_output_size/block_size.x+1, maxpool_output_size/block_size.y+1);
    const dim3 num_of_blocks(input_size/block_size.x+1, input_size/block_size.y+1);

    // memory allocation for the device
    float *dev_mem_a, *dev_mem_b, *dev_mem_c, *dev_mem_input, *gemm_output, *maxpool_output;
    hipMalloc(&dev_mem_a, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_b, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_c, sizeof(float) * input_size * input_size);
    hipMalloc(&gemm_output, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_input, sizeof(float) * input_size * input_size);
    hipMalloc(&maxpool_output, sizeof(float) * maxpool_output_size * maxpool_output_size);
    
    // copy variable to device memory
    hipMemcpy(dev_mem_a, &a, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_b, &b, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_c, &c, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_input, &maxpool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);

    // launch CUDA kernels

    // First launch gemm kernel
    gemm<<<num_of_blocks, block_size>>>(dev_mem_a, dev_mem_b, dev_mem_c, alpha, beta, gemm_output, input_size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // Then run maxpooling
    maxpool<<<num_of_maxpool_blocks, block_size>>>(dev_mem_input, maxpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // allocate output buf in main memory
    float *gemm_output_buf = (float*) malloc (sizeof(float)*input_size*input_size);
    float *maxpool_output_buf = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    
    // copy results from device to host
    hipMemcpy(gemm_output_buf, gemm_output, sizeof(float)*input_size*input_size, hipMemcpyDeviceToHost);
    hipMemcpy(maxpool_output_buf, maxpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    
    // prints the results
    cout<<"\n========== GEMM OUTPUT ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<gemm_output_buf[i]<<" ";
    }
    cout<<"\n========== MAXPOOL OUTPUT ==========\n";
    for (int i = 0; i < maxpool_output_size * maxpool_output_size; ++i) {
        if(i%maxpool_output_size==0) cout<<"\n";
        cout<<maxpool_output_buf[i]<<" ";
    }
    cout<<'\n';

    hipFree(dev_mem_a);
    hipFree(dev_mem_b);
    hipFree(dev_mem_c);
    hipFree(gemm_output);
    hipFree(dev_mem_input);
    hipFree(maxpool_output);
    free(gemm_output_buf);
    free(maxpool_output_buf);
    return 0;
}
